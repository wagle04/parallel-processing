

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>


// function to initialize the matrix with randm value
void initializeMatrix(int* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            if (i != j) {
                // only assign random values to non-diagonal elements
                matrix[i * size + j] = rand() % 100 + 1; 
            } else {
                // assign 0 to diagonal elements
                matrix[i * size + j] = 0; 
            }
        }
    }
}

// function to print the matrix
void printMatrix(int* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            std::cout << matrix[i * size + j] << " ";
        }
        std::cout << std::endl;
    }
}

/* function for CUDA kernel that will be executed by each thread in parallel
performs the updates to the matrix using the Floyd-Warshall algorithm for a specific value of k
*/
__global__ void floydWarshall(int *D, int *vbuf, int *hbuf, int n, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {
        vbuf[i] = D[i * n + k];
        hbuf[j] = D[k * n + j];

        // ensure all threads finish updatting vbuf and hbuf before moving to the next iteration
        __syncthreads(); 

        // update the matrix D by shortesst path distance between vertices i and j
        D[i * n + j] = min(D[i * n + j], vbuf[i] + hbuf[j]);
    }
}

int main() {
    // matrix size
    // we can change the matrix size to 100, 1000 and son on in the line below
    const int n = 10;

    // initialize necessary variables
    int *h_D, *d_D, *h_vbuf, *d_vbuf, *h_hbuf, *d_hbuf;

    // allocate and initialize matrix D on host
    h_D = (int*)malloc(n * n * sizeof(int));
    initializeMatrix(h_D, n);

    // allocate and initialize vbuf and hbuf on host
    h_vbuf = (int*)malloc(n * sizeof(int));
    h_hbuf = (int*)malloc(n * sizeof(int));

    // print the original matrix
    std::cout << "Original Matrix:" << std::endl;
    printMatrix(h_D, n);

    // allocate matrix D on device
    hipMalloc((void**)&d_D, n * n * sizeof(int));

    // allocate vbuf and hbuf on device
    hipMalloc((void**)&d_vbuf, n * sizeof(int));
    hipMalloc((void**)&d_hbuf, n * sizeof(int));

    // copy matrix D from host to device
    hipMemcpy(d_D, h_D, n * n * sizeof(int), hipMemcpyHostToDevice);

    // set up grid and block dimensions
    dim3 blockDim(16, 16); 
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);


    for (int k = 0; k < n; ++k) {
        // launch kernel for parallel computation
        floydWarshall<<<gridDim, blockDim>>>(d_D, d_vbuf, d_hbuf, n, k);

        // ensure all threads finish before moving to the next iteration
        hipDeviceSynchronize(); 
    }

    // copy matrix D from device to host
    hipMemcpy(h_D, d_D, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // print the result matrix
    std::cout << "\nResult Matrix:" << std::endl;
    printMatrix(h_D, n);

    // free allocated memory
    free(h_D);
    free(h_vbuf);
    free(h_hbuf);
    hipFree(d_D);
    hipFree(d_vbuf);
    hipFree(d_hbuf);

    return 0;
}